
#include <hip/hip_runtime.h>
extern "C" __global__
void resizeKernel(unsigned char* input, int inWidth, int inHeight, unsigned char* output, int outWidth, int outHeight) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < outWidth && y < outHeight) {
        int srcX = (x * inWidth) / outWidth;
        int srcY = (y * inHeight) / outHeight;
        int srcIdx = (srcY * inWidth + srcX) * 4; // Assuming RGBA
        int dstIdx = (y * outWidth + x) * 4;

        output[dstIdx] = input[srcIdx];      // R
        output[dstIdx + 1] = input[srcIdx + 1];  // G
        output[dstIdx + 2] = input[srcIdx + 2];  // B
        output[dstIdx + 3] = input[srcIdx + 3];  // A
    }
}
